
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <algorithm>
#include <map>
#include <chrono>
// nvcc -rdc=true .\rewrite.cu -lcudadevrt -o rewrite
// .\rewrite.exe transactional_T10I4D10K.csv 10 a.txt
// output 504
__device__ int patCount = 0;

struct TreeNode
{
    int item;
    int count;
    int childrenCount;

    // parent node
    TreeNode *parent;
    TreeNode **children;

    // __host__ __device__ TreeNode(int item, int count, TreeNode *parent)
    // __host__ __device__ ~TreeNode()
    // __host__ __device__ void addChild(int item, int count, int &location)
};

struct KV
{
    int item;
    TreeNode **itemLocations;
    int itemLocationsCount;

    // __host__ __device__ KV(int item, TreeNode *itemLocation)
    // __host__ __device__ ~KV()
    // __host__ __device__ void addItemLocation(TreeNode *itemLocation)
};

struct dMap
{
    int numOfItems;
    KV **keyVal;

    // __host__ __device__ dMap()
    // __host__ __device__ ~dMap()
    // __host__ __device__ int find(int item)
    // __host__ __device__ void add(TreeNode *itemLocation)
};

__device__ TreeNode *makeNode(int item, int count, TreeNode *parent)
{
    TreeNode *node = (TreeNode *)malloc(sizeof(TreeNode));
    node->item = item;
    node->count = count;
    node->parent = parent;
    node->childrenCount = 0;
    node->children = (TreeNode **)malloc(sizeof(TreeNode *) * count);
    return node;
}

__device__ dMap *makeMap()
{
    dMap *map = (dMap *)malloc(sizeof(dMap));
    map->numOfItems = 0;
    map->keyVal = (KV **)malloc(sizeof(KV *) * map->numOfItems);
    return map;
}

__device__ int findItem(dMap *map, int item)
{
    for (int i = 0; i < map->numOfItems; i++)
    {
        if (map->keyVal[i]->item == item)
        {
            return i;
        }
    }
    return -1;
}

__device__ void addNodeToKV(KV *keyVal, TreeNode *node)
{
    TreeNode **itemLocations = (TreeNode **)malloc(sizeof(TreeNode *) * keyVal->itemLocationsCount+1);
    for (int i = 0; i < keyVal->itemLocationsCount; i++)
    {
        itemLocations[i] = keyVal->itemLocations[i];
    }
    itemLocations[keyVal->itemLocationsCount] = node;
    keyVal->itemLocationsCount++;
    keyVal->itemLocations = itemLocations;
}

__device__ void addItemMap(dMap *map, TreeNode *node)
{
    // if item is not in map, add it
    int itemIndex = findItem(map, node->item);
    if (itemIndex == -1)
    {
        // new map
        KV **newKeyVal = (KV **)malloc(sizeof(KV *) * (map->numOfItems + 1));
        for (int i = 0; i < map->numOfItems; i++)
        {
            newKeyVal[i] = map->keyVal[i];
        }
        newKeyVal[map->numOfItems] = (KV *)malloc(sizeof(KV));
        newKeyVal[map->numOfItems]->item = node->item;
        newKeyVal[map->numOfItems]->itemLocationsCount = 1;
        newKeyVal[map->numOfItems]->itemLocations = (TreeNode **)malloc(sizeof(TreeNode *) * 1);
        newKeyVal[map->numOfItems]->itemLocations[0] = node;
        map->numOfItems++;
        map->keyVal = newKeyVal;
    }
    else
    {
        // add node to existing map
        addNodeToKV(map->keyVal[itemIndex], node);
    }

}

__device__ int *getPath(TreeNode *node)
{
    int nodes = 1;
    int *path = (int *)malloc(sizeof(int) * nodes);
    path[0] = nodes - 1;
    while (node->parent->item != -1)
    {
        nodes++;
        int *temp = (int *)malloc(sizeof(int) * nodes);
        for (int i = 0; i < nodes - 1; i++)
        {
            temp[i] = path[i];
        }
        temp[nodes - 1] = node->parent->item;
        path = temp;
        node = node->parent;
    }
    // printf("%d\n", nodes);
    path[0] = nodes - 1;

    return path;
}

__device__ void addChild(TreeNode *node, int item, int count, int &location, dMap *top)
{
    if (node->childrenCount == 0)
    {
        node->childrenCount = 1;
        TreeNode *child = makeNode(item, count, node);
        node->children = (TreeNode **)malloc(sizeof(TreeNode *) * count);
        node->children[0] = child;
        location = 0;
        addItemMap(top, child);
        return;
    }
    else
    {
        for (int i = 0; i < node->childrenCount; i++)
        {
            if (node->children[i]->item == item)
            {
                node->children[i]->count += count;
                location = i;
                return;
            }
        }
        node->childrenCount++;
        TreeNode *child = makeNode(item, count, node);
        TreeNode **newChildren = (TreeNode **)malloc(sizeof(TreeNode *) * node->childrenCount);
        for (int i = 0; i < node->childrenCount - 1; i++)
        {
            newChildren[i] = node->children[i];
        }
        newChildren[node->childrenCount - 1] = child;
        node->children = newChildren;
        location = node->childrenCount - 1;
        addItemMap(top, child);
        return;
    }
    return;
}

__global__ void partialMinerInitialize(dMap *map, int minSup);

__device__ void partialMiner(KV *keyVal, int minSup)
{
    // printf("Item: %d, depth: %d\n", keyVal->item, depth);
    dMap *map = makeMap();
    TreeNode *root = makeNode(-1,0,NULL);
    for (int i = 0; i < keyVal->itemLocationsCount; i++)
    {
        TreeNode *node = keyVal->itemLocations[i];
        TreeNode *partial = root;
        int location = -1;
        int *path = getPath(node);
        // printf("Nodes in path: %d", path[0]);
        for (int j = 0; j < path[0]; j++)
        {
            int item = path[j + 1];
            // printf("%d ", item);
            addChild(partial, item, node->count, location, map);
            partial = partial->children[location];
        }
        // printf("\n");
    }
    int grid = map->numOfItems / 1024 + 1;

    partialMinerInitialize<<<grid,1024>>>(map, minSup);
    return;
    // partialMinerInitialize(map, minSup, depth);
}

__global__ void partialMinerInitialize(dMap *map, int minSup)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < map->numOfItems)
    // for (int tid = 0; tid < map->numOfItems; tid++)
    {
        printf("Item: %d, TID: %d\n", map->keyVal[tid]->item, tid);
        KV *keyVal = map->keyVal[tid];
        int count = 0;
        for (int i = 0; i < keyVal->itemLocationsCount; i++)
        {
            count += keyVal->itemLocations[i]->count;
        }

        if (count >= minSup)
        {
            // printf("Pattern Found\n");
            // printf("Item %d has support %d\n", keyVal->item, count);
            // patCount[tid] = count;
            // patCount++;
            // printf("Pattern Count: %d\n", patCount);
            // __syncthreads();
            // __threadfence_system();
            // atomicAdd(&patCount, 1);
            // __threadfence_system();
            // __syncthreads();
            // printf("Pattern Count: %d\n", patCount);
            partialMiner(keyVal, minSup);
        }
        return;

    }
    return;
}

__global__ void buildInitialTree(int *transactions, int *indexes, int numOfIndexes, int minSup)
{
    TreeNode *root = makeNode(-1, 0, NULL);
    dMap *top = makeMap();
    patCount = 0;

    // printf("Constructing tree\n");

    for (int i = 0; i < numOfIndexes; i++)
    {
        TreeNode *current = root;
        // printf("Num of root children: %d\n", root->childrenCount);

        for (int j = indexes[i]; j < indexes[i + 1]; j++)
        {
            int item = transactions[j];
            // printf("%d ", item);
            int count = 1;
            int location = -1;
            addChild(current, item, count, location, top);
            current = current->children[location];
        }
        
    }

    // printf("Finished constructing tree\n");
    int grid = top->numOfItems / 1024 + 1;

    // print all items in map
    partialMinerInitialize<<<grid,1024>>>(top, minSup);
    // cudaDeviceSynchronize();

    // partialMinerInitialize(top, minSup, 0);
    // printf("Num of patterns: %d\n", patCount);

    // printf("Finished mining\n");
    return;
}

__global__ void printFinalPatterns()
{
    printf("Num of patterns: %d\n", patCount);
}

void initialTree(std::string fileName, int minimumSupport)
{
    std::map<int, int> map;
    std::ifstream file(fileName);
    std::string line;

    std::vector<std::vector<int>> transactions;
    while (std::getline(file, line))
    {
        std::stringstream ss(line);
        int item;
        std::vector<int> transaction;
        while (ss >> item)
        {
            transaction.push_back(item);
            map[item]++;
        }

        transactions.push_back(transaction);
    }

    std::vector<std::pair<int, int>> vec;
    for (auto it = map.begin(); it != map.end(); it++)
    {
        vec.push_back(*it);
    }

    std::sort(vec.begin(), vec.end(), [](const std::pair<int, int> &a, const std::pair<int, int> &b)
              { return a.second > b.second; });

    for (int i = 0; i < vec.size(); i++)
    {
        if (vec[i].second < minimumSupport)
        {
            vec.erase(vec.begin() + i);
            i--;
        }
    }

    std::vector<int> keys;
    for (auto it = vec.begin(); it != vec.end(); it++)
    {
        keys.push_back(it->first);
    }

    int *indexes = new int[transactions.size() + 1];
    indexes[0] = 0;
    int sumOfTransactions = 0;
    for (int i = 0; i < transactions.size(); i++)
    {
        sumOfTransactions += transactions[i].size();

        for (int j = 0; j < transactions[i].size(); j++)
        {
            if (std::find(keys.begin(), keys.end(), transactions[i][j]) == keys.end())
            {
                transactions[i].erase(transactions[i].begin() + j);
                j--;
            }
        }

        // sort transaction using keys
        std::sort(transactions[i].begin(), transactions[i].end(), [&keys](int a, int b)
                  { return std::find(keys.begin(), keys.end(), a) < std::find(keys.begin(), keys.end(), b); });
        indexes[i + 1] = indexes[i] + transactions[i].size();
    }

    int *flattenedTransactions = new int[sumOfTransactions];
    int index = 0;
    for (int i = 0; i < transactions.size(); i++)
    {
        for (int j = 0; j < transactions[i].size(); j++)
        {
            flattenedTransactions[index] = 0;
            flattenedTransactions[index] = transactions[i][j];
            index++;
        }
    }

    int *devFlattenedTransactions;
    int *devIndexes;

    // set heap size to 1GB
    hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);

    hipMalloc((void **)&devFlattenedTransactions, sizeof(int) * sumOfTransactions);
    hipMalloc((void **)&devIndexes, sizeof(int) * (transactions.size() + 1));

    hipMemcpy(devFlattenedTransactions, flattenedTransactions, sizeof(int) * sumOfTransactions, hipMemcpyHostToDevice);
    hipMemcpy(devIndexes, indexes, sizeof(int) * (transactions.size() + 1), hipMemcpyHostToDevice);

    buildInitialTree<<<1, 1>>>(devFlattenedTransactions, devIndexes, transactions.size(), minimumSupport);
    // buildInitialTree(flattenedTransactions, indexes, transactions.size(), minimumSupport);
    hipDeviceSynchronize();
    printFinalPatterns<<<1, 1>>>();
    hipDeviceSynchronize();
    // cudaFree(devFlattenedTransactions);
    // cudaFree(devIndexes);
}

int main(int argc, char *argv[])
{
    if (argc < 4)
    {
        printf("Usage: %s {file} {minimumSupport} {outputfile}\n", argv[0]);
        return 0;
    }

    // start time
    auto start = std::chrono::high_resolution_clock::now();

    std::string file = argv[1];
    int minimumSupport = atoi(argv[2]);

    initialTree(file, minimumSupport);

    auto endTime = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = endTime - start;
    printf("Time(seconds): %f\n", diff.count());

    return 0;
}