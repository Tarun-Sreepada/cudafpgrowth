// usage:
// nvcc .\fp.cu -o fp
// fp.exe {file} {minimumSupport} {outputFile}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <map>
#include <algorithm>
#include <chrono>

std::vector<std::pair<std::vector<int>, int>> patterns;

// general tree struct
struct TreeNode
{
    int item;
    int count;

    TreeNode *parent;
    std::vector<TreeNode *> children;

    TreeNode(int item, int count, TreeNode *parent)
    {
        this->item = item;
        this->count = count;
        this->parent = parent;
    }

    ~TreeNode()
    {
        for (int i = 0; i < children.size(); i++)
        {
            delete children[i];
        }
    }

    void addChild(TreeNode *child)
    {
        children.push_back(child);
    }

    TreeNode *find(int item)
    {
        TreeNode *current = this;
        for (int i = 0; i < children.size(); i++)
        {
            current = children[i];
            if (current->item == item)
            {
                return current;
            }
            current = this;
        }
        return NULL;
    }

    std::pair<std::vector<int>, int> getPath()
    {
        std::vector<int> path;
        int c = count;
        TreeNode *node = this;
        while (node->parent->item != -1)
        {
            path.push_back(node->parent->item);
            node = node->parent;
        }

        std::reverse(path.begin(), path.end());

        return std::make_pair(path, c);
    }

    void deleteTree()
    {
        for (int i = 0; i < children.size(); i++)
        {
            children[i]->deleteTree();
            delete children[i];
        }
        delete this;
    }
};

std::pair<std::vector<int>, std::vector<std::vector<int>>> mapForSorting(std::string fileName, int minimumSupport)
{
    std::map<int, int> map;
    std::ifstream file(fileName);
    std::string line;

    std::vector<std::vector<int>> transactions;
    while (std::getline(file, line))
    {
        std::stringstream ss(line);
        int item;
        std::vector<int> transaction;
        while (ss >> item)
        {
            transaction.push_back(item);
            map[item]++;
        }

        transactions.push_back(transaction);
    }

    // sort map
    std::vector<std::pair<int, int>> vec;
    for (auto it = map.begin(); it != map.end(); it++)
    {
        vec.push_back(*it);
    }

    std::sort(vec.begin(), vec.end(), [](const std::pair<int, int> &a, const std::pair<int, int> &b)
              { return a.second > b.second; });

    // remove items with support less than minimum support
    for (int i = 0; i < vec.size(); i++)
    {
        if (vec[i].second < minimumSupport)
        {
            vec.erase(vec.begin() + i);
            i--;
        }
    }

    // get keys
    std::vector<int> keys;
    for (auto it = vec.begin(); it != vec.end(); it++)
    {
        keys.push_back(it->first);
    }

    // remove items that are not in the keys
    for (auto it = transactions.begin(); it != transactions.end(); it++)
    {
        for (int i = 0; i < (*it).size(); i++)
        {
            if (std::find(keys.begin(), keys.end(), (*it)[i]) == keys.end())
            {
                (*it).erase((*it).begin() + i);
                i--;
            }
        }

        std::sort((*it).begin(), (*it).end(), [&keys](int a, int b)
                  { return std::find(keys.begin(), keys.end(), a) < std::find(keys.begin(), keys.end(), b); });
    }

    // return map
    return std::make_pair(keys, transactions);
}

std::pair<TreeNode *, std::map<int, std::vector<TreeNode *>>> buildTree(std::vector<int> keys, std::vector<std::vector<int>> vec)
{
    std::map<int, std::vector<TreeNode *>> map;
    TreeNode *root = new TreeNode(-1, 0, NULL);

    int nodes = 0;
    for (int i = 0; i < vec.size(); i++)
    {
        TreeNode *node = root;
        for (int j = 0; j < vec[i].size(); j++)
        {
            // convert item to int
            int item = vec[i][j];

            TreeNode *child = node->find(item);

            // if child doesn't exist, create it
            if (child == NULL)
            {
                nodes++;
                child = new TreeNode(item, 1, node);
                node->addChild(child);
                // if item is not in the map, create a new array and point to it
                if (map.find(item) == map.end())
                {
                    std::vector<TreeNode *> arr;
                    arr.push_back(child);
                    map[item] = arr;
                }
                // if item is in the map, add child to the array
                else
                {
                    map[item].push_back(child);
                }
            }
            else
            {
                child->count++;
            }
            node = child;
        }
    }

    return std::make_pair(root, map);
}

std::pair<TreeNode *, std::map<int, std::vector<TreeNode *>>> buildVariableTree(std::vector<int> keys, std::vector<std::pair<std::vector<int>, int>> vec)
{
    std::map<int, std::vector<TreeNode *>> map;
    TreeNode *root = new TreeNode(-1, 0, NULL);

    for (int i = 0; i < vec.size(); i++)
    {
        TreeNode *node = root;
        for (int j = 0; j < vec[i].first.size(); j++)
        {
            // convert item to int
            int item = vec[i].first[j];

            TreeNode *child = node->find(item);

            // if child doesn't exist, create it
            if (child == NULL)
            {
                child = new TreeNode(item, vec[i].second, node);
                node->addChild(child);
                // if item is not in the map, create a new array and point to it
                if (map.find(item) == map.end())
                {
                    std::vector<TreeNode *> arr;
                    arr.push_back(child);
                    map[item] = arr;
                }
                // if item is in the map, add child to the array
                else
                {
                    map[item].push_back(child);
                }
            }
            else
            {
                child->count += vec[i].second;
            }
            node = child;
        }
    }

    return std::make_pair(root, map);
}

void printTree(TreeNode *root)
{
    std::vector<int> path = root->getPath().first;
    int count = root->getPath().second;
    std::cout << "(";
    for (int i = 0; i < path.size(); i++)
    {
        std::cout << path[i];
        if (i != path.size() - 1)
        {
            std::cout << " ";
        }
    }
    std::cout << ") " << count << std::endl;
    for (int i = 0; i < root->children.size(); i++)
    {
        printTree(root->children[i]);
    }
}

// tree, dict, sorting keys, base, minimum support
void mining(TreeNode *root, std::map<int, std::vector<TreeNode *>> map, std::vector<int> keys, std::vector<int> base, int minimumSupport)
{
    // base copy
    std::vector<int> baseCopy = base;
    std::vector<std::pair<std::vector<int>, int>> vec;

    for (auto it = map.begin(); it != map.end(); it++)
    {
        int count = 0;
        for (int i = 0; i < it->second.size(); i++)
        {
            count += it->second[i]->count;
            vec.push_back(std::make_pair(it->second[i]->getPath().first, it->second[i]->count));
        }

        // if count is greater than minimum support
        if (count >= minimumSupport)
        {
            // add item to base
            base.push_back(it->first);

            patterns.push_back(std::make_pair(base, count));

            // build variable tree
            std::pair<TreeNode *, std::map<int, std::vector<TreeNode *>>> tree = buildVariableTree(base, vec);
            mining(tree.first, tree.second, keys, base, minimumSupport);
            // tree.first->deleteTree();
        }

        vec.clear();
        base = baseCopy;
    }
}

int main(int argc, char *argv[])
{
    if (argc < 4)
    {
        printf("Usage: %s {file} {minimumSupport} {outputfile}\n", argv[0]);
        return 0;
    }

    // start time
    auto start = std::chrono::high_resolution_clock::now();

    std::string file = argv[1];
    int minimumSupport = atoi(argv[2]);

    // make hashmap
    std::pair<std::vector<int>, std::vector<std::vector<int>>> map = mapForSorting(file, minimumSupport);
    std::vector<int> keys = map.first;
    std::vector<std::vector<int>> transactions = map.second;

    // make tree
    std::pair<TreeNode *, std::map<int, std::vector<TreeNode *>>> tree = buildTree(keys, transactions);
    TreeNode *root = tree.first;
    std::map<int, std::vector<TreeNode *>> map2 = tree.second;

    std::vector<int> base;
    for (auto it = map2.begin(); it != map2.end(); it++)
    {
        // std::cout << it->first << std::endl;
        int count = 0;
        std::vector<std::pair<std::vector<int>, int>> vec;
        for (int i = 0; i < it->second.size(); i++)
        {
            count += it->second[i]->count;
            vec.push_back(std::make_pair(it->second[i]->getPath().first, it->second[i]->count));
        }

        // if count is greater than minimum support
        // std::cout << "count: " << count << std::endl;
        if (count >= minimumSupport)
        {
            // add item to base
            base.push_back(it->first);
            // add item to patterns
            patterns.push_back(std::make_pair(base, count));
            // build variable tree
            std::pair<TreeNode *, std::map<int, std::vector<TreeNode *>>> tree = buildVariableTree(base, vec);
            mining(tree.first, tree.second, keys, base, minimumSupport);
        }

        // empty base
        base.clear();
    }

    // delete tree
    // root->deleteTree();

    // end time
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;

    // write to file
    std::ofstream outfile;
    outfile.open(argv[3]);
    outfile << "Time: " << diff.count() << " seconds" << std::endl;
    outfile << "Number of patterns: " << patterns.size() << std::endl;
    for (auto it = patterns.begin(); it != patterns.end(); it++)
    {
        for (int i = 0; i < it->first.size(); i++)
        {
            outfile << it->first[i];
            if (i != it->first.size() - 1)
            {
                outfile << " ";
            }
        }
        outfile << ": " << it->second << std::endl;
    }

    std::cout << "Time: " << diff.count() << " seconds" << std::endl;
    std::cout << "Number of patterns: " << patterns.size() << std::endl;

    return 0;
}